
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define S_TO_NSEC 1000000000

__global__
void access(float* A, long long n) {
    A[0] = A[n-1];
}

int main(int argc, char *argv[]) {
    // Track synchronous and asynchronous errors
    hipError_t syncErr, asyncErr;

    // Get length of array
    long long n = strtol(argv[1], NULL, 10); // is long long to avoid overflow
    assert(n>=2);

    // Allocate memory, accessible by CPU and GPU
    float *A;
    hipMallocManaged(&A, n * sizeof(float));
    syncErr = hipGetLastError();
    if (syncErr != hipSuccess) printf("syncErr: %s\n", hipGetErrorString(syncErr));

    // Set last element of array
    float last = -84.845;
    A[n-1] = last;

    // Time taken to run GPU kernel with 1 block and 1 thread in it
    struct timespec start;
    struct timespec end;

    clock_gettime(CLOCK_MONOTONIC, &start);
    access<<<1, 1>>>(A, n);
    asyncErr = hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);
    if (asyncErr != hipSuccess) printf("syncErr: %s\n", hipGetErrorString(asyncErr));

    // Print timings
    time_t sec = end.tv_sec - start.tv_sec;
    long nsec = end.tv_nsec - start.tv_nsec;
    printf("%lld,%ld\n", n, sec * S_TO_NSEC + nsec);

    // Check successfully changed first element
    assert(A[0] == last);

    // Free memory
    hipFree(A);
}
